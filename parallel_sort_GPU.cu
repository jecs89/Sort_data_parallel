#include <iostream>
#include <vector>
#include <ctime>
#include <cstdlib>
#include <boost/chrono.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

//Fill a vector with random numbers in the range [lower, upper]
void rnd_fill(thrust::host_vector<double> &V, const double lower, const double upper, const unsigned int seed) {

    //Create a unique seed for the random number generator
    srand(time(NULL));
    
    size_t elem = V.size();
    for( size_t i = 0; i < elem; ++i){
        V[i] = (double) rand() / (double) RAND_MAX;
    }
}



int main() {
    thrust::host_vector<double> V;
    thrust::device_vector<double> d_V;
    
    //use the system time to create a random seed
    unsigned int seed = (unsigned int) time(NULL);
    
    size_t step = 10;
    size_t mem = 10000000;

    for(size_t i = 16; i <= mem; i = 2 * step, step *= 1.1) {
        //Fill V with random numbers in the range [0,1]:
        V.resize(i);
        rnd_fill(V, 0.0, 1.0, seed);
        
        boost::chrono::steady_clock::time_point start_cpu = boost::chrono::steady_clock::now();
        d_V = V; // Transfer data to the GPU
        boost::chrono::steady_clock::time_point end_cpu = boost::chrono::steady_clock::now();
        double dt1 = boost::chrono::duration <double, boost::milli> (end_cpu - start_cpu).count();

	    hipEvent_t start, stop;
	    hipEventCreate(&start);
	    hipEventCreate(&stop);

	    //Start recording
	    hipEventRecord(start,0);
        
            thrust::stable_sort(d_V.begin(), d_V.end());
        
	    //Stop recording
	    hipEventRecord(stop,0);
	    hipEventSynchronize(stop);
	    float elapsedTime;
	    hipEventElapsedTime(&elapsedTime, start, stop);

	    hipEventDestroy(start);
	    hipEventDestroy(stop);
	    
        start_cpu = boost::chrono::steady_clock::now();
        V = d_V; // Transfer data to the CPU
        end_cpu = boost::chrono::steady_clock::now();
        double dt2 = boost::chrono::duration <double, boost::milli> (end_cpu - start_cpu).count();
	    

	    //std::cout << i << "\t" << elapsedTime << "\t" << dt1 + dt2 << std::endl;
	    std::cout << i << "\t" << elapsedTime + dt1 + dt2 << std::endl;
    }
    
    return 0;
}

